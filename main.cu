
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <regex>
#include <fstream>
#include <unordered_map>
#include <unordered_set>
#include <map>
#include <chrono>
using namespace std;
class DB{
public:
    vector<vector<int>> item,tid;
    vector<vector<int>> iu,ru;
    vector<int> sequence_len;

    vector<int> SWUtility;

    int DButility=0;

    unordered_map<int,int> item_swu;
};

class GPU_DB {
public:
    int **item,**tid;
    int **iu,**ru;

    int *sequence_len;
    int sid_len;
};

void parseData(ifstream &file,DB &DBdata) {
    regex numberRegex("(\\d+)\\[(\\d+)\\]");

    vector<int> item,tid;
    vector<int> iu,ru;

    string line;
    while (getline(file, line)) {
        istringstream iss(line);
        int Tid = 1;
        int seq_len=0;
        string token;
        while (iss >> token) {
            if (token == "-1") {
                Tid++;
            } else if (token == "-2") {
                Tid = 1;

                DBdata.sequence_len.push_back(seq_len);
                seq_len=0;
            } else if (token.find("SUtility:") != string::npos) {
                int sUtility = stoi(token.substr(token.find(":") + 1));
                DBdata.SWUtility.push_back(sUtility);
                DBdata.DButility+=sUtility;
                unordered_set<int> uniqueSet(item.begin(),item.end());
                for(int i:uniqueSet){
                    if(DBdata.item_swu.find(i)!=DBdata.item_swu.end()){
                        DBdata.item_swu[i]+=sUtility;
                    }else{
                        DBdata.item_swu[i]=sUtility;
                    }
                }


                DBdata.item.push_back(item);
                DBdata.iu.push_back(iu);
                DBdata.ru.push_back(ru);
                DBdata.tid.push_back(tid);

                item.clear();
                iu.clear();
                ru.clear();
                tid.clear();
            } else {
                smatch match;
                if (regex_match(token, match, numberRegex)) {
                    int firstValue = stoi(match[1]);
                    int secondValue = stod(match[2]);
                    item.push_back(firstValue);
                    iu.push_back(secondValue);
                    ru.push_back(0);
                    tid.push_back(Tid);
                    seq_len++;
                }
            }
        }
    }

    int sid_len = int(DBdata.sequence_len.size());
    int ru_tmp;
    for(int i=0;i<sid_len;i++) {
        ru_tmp =0;
        for (int j = 0; j < DBdata.sequence_len[i]; j++) {
            ru_tmp +=DBdata.iu[i][j];
            DBdata.ru[i][j] =  DBdata.SWUtility[i] - ru_tmp;
        }
    }
}

void SWUpruning(double threshold,DB &DBdata){
    unordered_set<int> ItemSwuUnderThreshold;
    for(pair<int,int>i:DBdata.item_swu){
        if(i.second<threshold){
            ItemSwuUnderThreshold.insert(i.first);
        }
    }

    DB update_DB;
    vector<int> item,tid;
    vector<int> iu,ru;
    int seq_len;

    int sid_len = int(DBdata.sequence_len.size());
    for(int i=0;i<sid_len;i++){
        seq_len=0;
        for(int j=0;j<DBdata.sequence_len[i];j++){
            if(ItemSwuUnderThreshold.find(DBdata.item[i][j])!=ItemSwuUnderThreshold.end()){
                for(int k=0;k<seq_len;k++){
                    ru[k]-=DBdata.iu[i][j];
                }
            }else{
                item.push_back(DBdata.item[i][j]);
                tid.push_back(DBdata.tid[i][j]);
                iu.push_back(DBdata.iu[i][j]);
                ru.push_back(DBdata.ru[i][j]);
                seq_len++;
            }
        }


        if(!item.empty()){
            update_DB.sequence_len.push_back(seq_len);

            update_DB.item.push_back(item);
            update_DB.iu.push_back(iu);
            update_DB.ru.push_back(ru);
            update_DB.tid.push_back(tid);

            item.clear();
            iu.clear();
            ru.clear();
            tid.clear();
        }

    }
    //cout<<"";
    DBdata = update_DB;

}

void Bulid_GPU_DB(DB &DBdata,GPU_DB &Gpu_Db){
    Gpu_Db.sid_len=int(DBdata.sequence_len.size());
    Gpu_Db.sequence_len = new int[Gpu_Db.sid_len];

    Gpu_Db.item = new int*[Gpu_Db.sid_len];
    Gpu_Db.iu = new int*[Gpu_Db.sid_len];
    Gpu_Db.ru = new int*[Gpu_Db.sid_len];
    Gpu_Db.tid = new int*[Gpu_Db.sid_len];

    for(int i=0;i<Gpu_Db.sid_len;i++){
        Gpu_Db.sequence_len[i] = DBdata.sequence_len[i];


        Gpu_Db.item[i] = new int[Gpu_Db.sequence_len[i]];
        Gpu_Db.iu[i] = new int[Gpu_Db.sequence_len[i]];
        Gpu_Db.ru[i] = new int[Gpu_Db.sequence_len[i]];
        Gpu_Db.tid[i] = new int[Gpu_Db.sequence_len[i]];
        for(int j=0;j<Gpu_Db.sequence_len[i];j++){
            Gpu_Db.item[i][j]=DBdata.item[i][j];
            Gpu_Db.iu[i][j]=DBdata.iu[i][j];
            Gpu_Db.ru[i][j]=DBdata.ru[i][j];
            Gpu_Db.tid[i][j]=DBdata.tid[i][j];

//            cout<<Gpu_Db.item[i][j]<<" ";
//            cout<<Gpu_Db.iu[i][j]<<" ";
//            cout<<Gpu_Db.ru[i][j]<<" ";
//            cout<<Gpu_Db.tid[i][j]<<"\n";
        }

//        cout<<Gpu_Db.sequence_len[i]<<endl;

    }
//    cout<<"";
}

int main() {

    // 指定要讀取的檔案名稱
    string filename = "Yoochoose.txt";
    ifstream file(filename);
    vector<string> lines;

    // 檢查檔案是否成功開啟
    if (!file.is_open()) {
        cerr << "無法開啟檔案: " << filename << endl;
        return 1; // 返回錯誤代碼
    }

    DB DBdata;
    parseData(file,DBdata);

    file.close(); // 關閉檔案

    double threshold = 0.000024 * DBdata.DButility;

    auto start = std::chrono::high_resolution_clock::now();

    SWUpruning(threshold,DBdata);




    GPU_DB Gpu_Db;

    Bulid_GPU_DB(DBdata,Gpu_Db);

    size_t freeMem = 0;
    size_t totalMem = 0;

    // 獲取 GPU 的內存信息
    hipError_t status = hipMemGetInfo(&freeMem, &totalMem);

    if (status == hipSuccess) {
        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
    } else {
        cerr << "無法獲取內存信息，錯誤碼: " << hipGetErrorString(status) << endl;
    }


    int **d_item_project;
    hipMalloc(&d_item_project, Gpu_Db.sid_len * sizeof(int*));

    int **d_iu_project;
    hipMalloc(&d_iu_project, Gpu_Db.sid_len * sizeof(int*));

    int **d_ru_project;
    hipMalloc(&d_ru_project, Gpu_Db.sid_len * sizeof(int*));

    int **d_tid_project;
    hipMalloc(&d_tid_project, Gpu_Db.sid_len * sizeof(int*));

    // 主機上的指標陣列，用於存放每一行的 d_tmp 指標
    //cudaMemcpy(&d_item_project[i], &d_tmp, sizeof(int*), cudaMemcpyDeviceToDevice);不能這樣,因為d_item_project[i]在host不能讀取
    //要開主機指標的指標（陣列）存Device指標
    int **h_item_project = new int*[Gpu_Db.sid_len];
    int **h_iu_project = new int*[Gpu_Db.sid_len];
    int **h_ru_project = new int*[Gpu_Db.sid_len];
    int **h_tid_project = new int*[Gpu_Db.sid_len];


    for(int i=0;i<Gpu_Db.sid_len;i++){
        int *d_tmp;
        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * 1000 * sizeof(int));//開大空間分配
        hipMemcpy(d_tmp,Gpu_Db.item[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_item_project[i] = d_tmp;

        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * 1000 * sizeof(int));//開大空間分配
        hipMemcpy(d_tmp,Gpu_Db.iu[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_iu_project[i] = d_tmp;

        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * 1000 * sizeof(int));//開大空間分配
        hipMemcpy(d_tmp,Gpu_Db.ru[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_ru_project[i] = d_tmp;

        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * 1000 * sizeof(int));//開大空間分配
        hipMemcpy(d_tmp,Gpu_Db.tid[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_tid_project[i] = d_tmp;
    }

    hipMemcpy(d_item_project, h_item_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(d_iu_project, h_iu_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(d_ru_project, h_ru_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(d_tid_project, h_tid_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);


    //kernelfunction操作


    // 獲取 GPU 的內存信息
    status = hipMemGetInfo(&freeMem, &totalMem);

    if (status == hipSuccess) {
        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
    } else {
        cerr << "無法獲取內存信息，錯誤碼: " << hipGetErrorString(status) << endl;
    }

    for(int i=0;i<Gpu_Db.sid_len;i++) {
        hipFree(h_item_project[i]);
        hipFree(h_iu_project[i]);
        hipFree(h_ru_project[i]);
        hipFree(h_tid_project[i]);
    }

    delete[] h_item_project;
    delete[] h_iu_project;
    delete[] h_ru_project;
    delete[] h_tid_project;

    hipFree(d_item_project);
    hipFree(d_iu_project);
    hipFree(d_ru_project);
    hipFree(d_tid_project);


//    // 獲取 GPU 的內存信息
//    status = cudaMemGetInfo(&freeMem, &totalMem);
//
//    if (status == cudaSuccess) {
//        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
//        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
//    } else {
//        cerr << "無法獲取內存信息，錯誤碼: " << cudaGetErrorString(status) << endl;
//    }


//    for(int i=0;i<Gpu_Db.sid_len;i++) {
//        for (int j = 0; j < Gpu_Db.sequence_len[i]; j++) {
//            cout << Gpu_Db.item[i][j] << " ";
//            cout << Gpu_Db.iu[i][j] << " ";
//            cout << Gpu_Db.ru[i][j] << " ";
//            cout << Gpu_Db.tid[i][j] << "\n";
//        }
//        cout << Gpu_Db.sequence_len[i] << endl;
//    }
    auto end = std::chrono::high_resolution_clock::now();


    // 計算持續時間，並轉換為毫秒
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds" << std::endl;

    return 0;
}
