
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <regex>
#include <fstream>
#include <unordered_map>
#include <unordered_set>
#include <map>
#include <chrono>
using namespace std;
class DB{
public:
    vector<vector<int>> item,tid;
    vector<vector<int>> iu,ru;
    vector<int> sequence_len;

    vector<int> SWUtility;

    int DButility=0;

    unordered_map<int,int> item_swu;
};

class GPU_DB {
    int **item,**tid;
    int **iu,**ru;

    int *sequence_len;
    int sid_len;
};

void parseData(ifstream &file,DB &DBdata) {
    regex numberRegex("(\\d+)\\[(\\d+)\\]");

    vector<int> item,tid;
    vector<int> iu,ru;

    string line;
    while (getline(file, line)) {
        istringstream iss(line);
        int Tid = 1;
        int seq_len=0;
        string token;
        while (iss >> token) {
            if (token == "-1") {
                Tid++;
            } else if (token == "-2") {
                Tid = 1;

                DBdata.sequence_len.push_back(seq_len);
                seq_len=0;
            } else if (token.find("SUtility:") != string::npos) {
                int sUtility = stoi(token.substr(token.find(":") + 1));
                DBdata.SWUtility.push_back(sUtility);
                DBdata.DButility+=sUtility;
                unordered_set<int> uniqueSet(item.begin(),item.end());
                for(int i:uniqueSet){
                    if(DBdata.item_swu.find(i)!=DBdata.item_swu.end()){
                        DBdata.item_swu[i]+=sUtility;
                    }else{
                        DBdata.item_swu[i]=sUtility;
                    }
                }


                DBdata.item.push_back(item);
                DBdata.iu.push_back(iu);
                DBdata.ru.push_back(ru);
                DBdata.tid.push_back(tid);

                item.clear();
                iu.clear();
                ru.clear();
                tid.clear();
            } else {
                smatch match;
                if (regex_match(token, match, numberRegex)) {
                    int firstValue = stoi(match[1]);
                    int secondValue = stod(match[2]);
                    item.push_back(firstValue);
                    iu.push_back(secondValue);
                    ru.push_back(0);
                    tid.push_back(Tid);
                    seq_len++;
                }
            }
        }
    }

    int sid_len = int(DBdata.sequence_len.size());
    int ru_tmp;
    for(int i=0;i<sid_len;i++) {
        ru_tmp =0;
        for (int j = 0; j < DBdata.sequence_len[i]; j++) {
            ru_tmp +=DBdata.iu[i][j];
            DBdata.ru[i][j] =  DBdata.SWUtility[i] - ru_tmp;
        }
    }
}

void SWUpruning(double threshold,DB &DBdata){
    unordered_set<int> ItemSwuUnderThreshold;
    for(pair<int,int>i:DBdata.item_swu){
        if(i.second<threshold){
            ItemSwuUnderThreshold.insert(i.first);
        }
    }

    DB update_DB;
    vector<int> item,tid;
    vector<int> iu,ru;
    int seq_len;

    int sid_len = int(DBdata.sequence_len.size());
    for(int i=0;i<sid_len;i++){
        seq_len=0;
        for(int j=0;j<DBdata.sequence_len[i];j++){
            if(ItemSwuUnderThreshold.find(DBdata.item[i][j])!=ItemSwuUnderThreshold.end()){
                for(int k=0;k<seq_len;k++){
                    ru[k]-=DBdata.iu[i][j];
                }
            }else{
                item.push_back(DBdata.item[i][j]);
                tid.push_back(DBdata.tid[i][j]);
                iu.push_back(DBdata.iu[i][j]);
                ru.push_back(DBdata.ru[i][j]);
                seq_len++;
            }
        }


        if(!item.empty()){
            update_DB.sequence_len.push_back(seq_len);

            update_DB.item.push_back(item);
            update_DB.iu.push_back(iu);
            update_DB.ru.push_back(ru);
            update_DB.tid.push_back(tid);

            item.clear();
            iu.clear();
            ru.clear();
            tid.clear();
        }

    }
    //cout<<"";
    DBdata = update_DB;

}

int main() {

    // 指定要讀取的檔案名稱
    string filename = "YoochooseSamller.txt";
    ifstream file(filename);
    vector<string> lines;

    // 檢查檔案是否成功開啟
    if (!file.is_open()) {
        cerr << "無法開啟檔案: " << filename << endl;
        return 1; // 返回錯誤代碼
    }

    DB DBdata;
    parseData(file,DBdata);

    file.close(); // 關閉檔案


    double threshold = 0.03 * DBdata.DButility;



    auto start = std::chrono::high_resolution_clock::now();

    SWUpruning(threshold,DBdata);

    auto end = std::chrono::high_resolution_clock::now();


    // 計算持續時間，並轉換為毫秒
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds" << std::endl;


    GPU_DB Gpu_Db;

    return 0;
}
