
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <regex>
#include <fstream>
#include <unordered_map>
#include <unordered_set>
#include <map>
#include <stack>
#include <chrono>

using namespace std;
class DB{
public:
    vector<vector<int>> item,tid;
    vector<vector<int>> iu,ru;
    vector<int> sequence_len;

    vector<int> SWUtility;

    int DButility=0;

    unordered_map<int,int> item_swu;
    vector<int> DB_item_set;
};

class GPU_DB {
public:
    int **item,**tid;
    int **iu,**ru;

    int *sequence_len;
    int max_sequence_len;

    int sid_len;

    vector<int> DB_item_set;
};
class Tree_node{
public:
    string pattern;
    int *ProjectArr_first_position,*ProjectArr_len;
    
};


//int test_max_seq=0;

void parseData(ifstream &file,DB &DBdata) {


    regex numberRegex("(\\d+)\\[(\\d+)\\]");

    vector<int> item,tid;
    vector<int> iu,ru;

    string line;
    while (getline(file, line)) {
        istringstream iss(line);
        int Tid = 1;
        int seq_len=0;
        string token;
        while (iss >> token) {
            if (token == "-1") {
                Tid++;
            } else if (token == "-2") {
                Tid = 1;

                DBdata.sequence_len.push_back(seq_len);
                seq_len=0;
            } else if (token.find("SUtility:") != string::npos) {
                int sUtility = stoi(token.substr(token.find(":") + 1));
                DBdata.SWUtility.push_back(sUtility);
                DBdata.DButility+=sUtility;
                unordered_set<int> uniqueSet(item.begin(),item.end());
                for(int i:uniqueSet){
                    if(DBdata.item_swu.find(i)!=DBdata.item_swu.end()){
                        DBdata.item_swu[i]+=sUtility;
                    }else{
                        DBdata.item_swu[i]=sUtility;
                    }
                }

//                if(item.size()>test_max_seq){
//                    test_max_seq=item.size();
//                }
                DBdata.item.push_back(item);
                DBdata.iu.push_back(iu);
                DBdata.ru.push_back(ru);
                DBdata.tid.push_back(tid);

                item.clear();
                iu.clear();
                ru.clear();
                tid.clear();
            } else {
                smatch match;
                if (regex_match(token, match, numberRegex)) {
                    int firstValue = stoi(match[1]);
                    int secondValue = stod(match[2]);
                    item.push_back(firstValue);
                    iu.push_back(secondValue);
                    ru.push_back(0);
                    tid.push_back(Tid);
                    seq_len++;
                }
            }
        }
    }

    int sid_len = int(DBdata.sequence_len.size());
    int ru_tmp;
    for(int i=0;i<sid_len;i++) {
        ru_tmp =0;
        for (int j = 0; j < DBdata.sequence_len[i]; j++) {
            ru_tmp +=DBdata.iu[i][j];
            DBdata.ru[i][j] =  DBdata.SWUtility[i] - ru_tmp;
        }
    }
}

void SWUpruning(double threshold,DB &DBdata){
    DB update_DB;

//    unordered_set<int> ItemSwuUnderThreshold;
//    for(pair<int,int>i:DBdata.item_swu){
//        if(i.second<threshold){
//            ItemSwuUnderThreshold.insert(i.first);
//        }else{
//            update_DB.DB_item_set.push_back(i.first);
//        }
//    }
    vector<int> item,tid;
    vector<int> iu,ru;
    int seq_len;

    int sid_len = int(DBdata.sequence_len.size());
    for(int i=0;i<sid_len;i++){
        seq_len=0;
        for(int j=0;j<DBdata.sequence_len[i];j++){
            if(DBdata.item_swu[DBdata.item[i][j]]<threshold){
                for(int k=0;k<seq_len;k++){
                    ru[k]-=DBdata.iu[i][j];
                }
            }else{
                item.push_back(DBdata.item[i][j]);
                tid.push_back(DBdata.tid[i][j]);
                iu.push_back(DBdata.iu[i][j]);
                ru.push_back(DBdata.ru[i][j]);
                seq_len++;
            }
        }


        if(!item.empty()){
            update_DB.sequence_len.push_back(seq_len);

            update_DB.item.push_back(item);
            update_DB.iu.push_back(iu);
            update_DB.ru.push_back(ru);
            update_DB.tid.push_back(tid);

            item.clear();
            iu.clear();
            ru.clear();
            tid.clear();
        }

    }
    //cout<<"";
    DBdata = update_DB;

}

void Bulid_GPU_DB(DB &DBdata,GPU_DB &Gpu_Db){
    Gpu_Db.sid_len=int(DBdata.sequence_len.size());
    Gpu_Db.sequence_len = new int[Gpu_Db.sid_len];

    Gpu_Db.item = new int*[Gpu_Db.sid_len];
    Gpu_Db.iu = new int*[Gpu_Db.sid_len];
    Gpu_Db.ru = new int*[Gpu_Db.sid_len];
    Gpu_Db.tid = new int*[Gpu_Db.sid_len];

    int max_sequence_len=0;

    for(int i=0;i<Gpu_Db.sid_len;i++){
        Gpu_Db.sequence_len[i] = DBdata.sequence_len[i];

        if(Gpu_Db.sequence_len[i]>max_sequence_len){
            max_sequence_len=Gpu_Db.sequence_len[i];
        }


        Gpu_Db.item[i] = new int[Gpu_Db.sequence_len[i]];
        Gpu_Db.iu[i] = new int[Gpu_Db.sequence_len[i]];
        Gpu_Db.ru[i] = new int[Gpu_Db.sequence_len[i]];
        Gpu_Db.tid[i] = new int[Gpu_Db.sequence_len[i]];
        for(int j=0;j<Gpu_Db.sequence_len[i];j++){
            Gpu_Db.item[i][j]=DBdata.item[i][j];
            Gpu_Db.iu[i][j]=DBdata.iu[i][j];
            Gpu_Db.ru[i][j]=DBdata.ru[i][j];
            Gpu_Db.tid[i][j]=DBdata.tid[i][j];

//            cout<<Gpu_Db.item[i][j]<<" ";
//            cout<<Gpu_Db.iu[i][j]<<" ";
//            cout<<Gpu_Db.ru[i][j]<<" ";
//            cout<<Gpu_Db.tid[i][j]<<"\n";
        }

//        cout<<Gpu_Db.sequence_len[i]<<endl;

    }
    Gpu_Db.max_sequence_len=max_sequence_len;

    Gpu_Db.DB_item_set=DBdata.DB_item_set;
    //cout<<max_sequence_len;
}

__global__ void PEUcounter(int project_item,
                           int** d_item_project, int** d_iu_project, int** d_ru_project, int** d_tid_project,
                           int* d_sequence_len,int sid_len,
                           int *d_PEU_seq,int *d_Utility_seq,int *d_project_point) {

    if(threadIdx.x<d_sequence_len[blockIdx.x]){
        if(d_item_project[blockIdx.x][threadIdx.x]==project_item){
            atomicMax(&d_PEU_seq[blockIdx.x], d_iu_project[blockIdx.x][threadIdx.x]+d_ru_project[blockIdx.x][threadIdx.x]);
            atomicMin(&d_project_point[blockIdx.x],int(blockIdx.x));
            atomicMax(&d_Utility_seq[blockIdx.x], d_iu_project[blockIdx.x][threadIdx.x]);
        }
    }
}

__global__ void PeuCounter(int project_item,
                           int** d_item_project, int** d_iu_project, int** d_ru_project, int** d_tid_project,
                           int* d_sequence_len,int sid_len,
                           int *d_PEU_seq,int *d_project_point) {

    if(threadIdx.x<d_sequence_len[blockIdx.x]){
        if(d_item_project[blockIdx.x][threadIdx.x]==project_item){
            atomicMax(&d_PEU_seq[blockIdx.x], d_iu_project[blockIdx.x][threadIdx.x]+d_ru_project[blockIdx.x][threadIdx.x]);
            atomicMin(&d_project_point[blockIdx.x],int(blockIdx.x));
        }
    }
}

int main() {
    auto start = std::chrono::high_resolution_clock::now();

    // 指定要讀取的檔案名稱
    string filename = "Yoochoose.txt";
    ifstream file(filename);
    vector<string> lines;

    // 檢查檔案是否成功開啟
    if (!file.is_open()) {
        cerr << "無法開啟檔案: " << filename << endl;
        return 1; // 返回錯誤代碼
    }


    DB DBdata;
    parseData(file,DBdata);

//    cout<<test_max_seq;
    file.close(); // 關閉檔案

    double threshold = 0.00014 * DBdata.DButility;

    SWUpruning(threshold,DBdata);

    GPU_DB Gpu_Db;

    Bulid_GPU_DB(DBdata,Gpu_Db);

    
    stack<Tree_node> Main_stack;
    Tree_node top1_Node;

    top1_Node.pattern = "";
    top1_Node.ProjectArr_first_position=new int[Gpu_Db.sid_len];
    top1_Node.ProjectArr_len=new int[Gpu_Db.sid_len];
    
    //DB clearDBdata;
    //DBdata = clearDBdata;
    //cout<<Gpu_Db.max_sequence_len;
    
    
    size_t freeMem = 0;
    size_t totalMem = 0;

    // 獲取 GPU 的內存信息
    hipError_t status = hipMemGetInfo(&freeMem, &totalMem);

    if (status == hipSuccess) {
        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
    } else {
        cerr << "無法獲取內存信息，錯誤碼: " << hipGetErrorString(status) << endl;
    }


    int **d_item_project;
    hipMalloc(&d_item_project, Gpu_Db.sid_len * sizeof(int*));

    int **d_iu_project;
    hipMalloc(&d_iu_project, Gpu_Db.sid_len * sizeof(int*));

    int **d_ru_project;
    hipMalloc(&d_ru_project, Gpu_Db.sid_len * sizeof(int*));

    int **d_tid_project;
    hipMalloc(&d_tid_project, Gpu_Db.sid_len * sizeof(int*));

    // 主機上的指標陣列，用於存放每一行的 d_tmp 指標
    //cudaMemcpy(&d_item_project[i], &d_tmp, sizeof(int*), cudaMemcpyDeviceToDevice);不能這樣,因為d_item_project[i]在host不能讀取
    //要開主機指標的指標（陣列）存Device指標
    int **h_item_project = new int*[Gpu_Db.sid_len];
    int **h_iu_project = new int*[Gpu_Db.sid_len];
    int **h_ru_project = new int*[Gpu_Db.sid_len];
    int **h_tid_project = new int*[Gpu_Db.sid_len];


    for(int i=0;i<Gpu_Db.sid_len;i++){
        int *d_tmp;
        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * 1000 * sizeof(int));//開大空間分配
        hipMemcpy(d_tmp,Gpu_Db.item[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_item_project[i] = d_tmp;

        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * 1000 * sizeof(int));//開大空間分配
        hipMemcpy(d_tmp,Gpu_Db.iu[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_iu_project[i] = d_tmp;

        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * 1000 * sizeof(int));//開大空間分配
        hipMemcpy(d_tmp,Gpu_Db.ru[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_ru_project[i] = d_tmp;

        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * 1000 * sizeof(int));//開大空間分配
        hipMemcpy(d_tmp,Gpu_Db.tid[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_tid_project[i] = d_tmp;
    }

    hipMemcpy(d_item_project, h_item_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(d_iu_project, h_iu_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(d_ru_project, h_ru_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(d_tid_project, h_tid_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);




    int *d_sequence_len;
    hipMalloc(&d_sequence_len, Gpu_Db.sid_len * sizeof(int));
    hipMemcpy(d_sequence_len,Gpu_Db.sequence_len,Gpu_Db.sid_len * sizeof(int),hipMemcpyHostToDevice);
//    for(int i=0;i<Gpu_Db.sid_len;i++){
//        cout<<Gpu_Db.sequence_len[i]<<endl;
//    }
//    cout<<endl;
    //kernelfunction操作

    //如果sequence長度有超過1024需要額外處理
    //int *block_per_sequence = new int[Gpu_Db.sid_len];
//    if(Gpu_Db.max_sequence_len>1024){
//        threadsPerBlock=1024;
//        for(int i=0;i<Gpu_Db.sid_len;i++){
//            blocksPerGrid+= (Gpu_Db.sequence_len[i]+threadsPerBlock-1)/threadsPerBlock;
//            block_per_sequence[i] = (Gpu_Db.sequence_len[i]+threadsPerBlock-1)/threadsPerBlock;
//        }
//    }else{
//        threadsPerBlock=Gpu_Db.max_sequence_len;
//        blocksPerGrid=Gpu_Db.sid_len;
//    }

    int *d_PEU_seq;
    hipMalloc(&d_PEU_seq, Gpu_Db.sid_len * sizeof(int));
    hipMemset(d_PEU_seq, 0, Gpu_Db.sid_len * sizeof(int));

    int *d_Utility_seq;
    hipMalloc(&d_Utility_seq, Gpu_Db.sid_len * sizeof(int));
    hipMemset(d_Utility_seq, 0, Gpu_Db.sid_len * sizeof(int));

    int *d_project_point;
    hipMalloc(&d_project_point, Gpu_Db.sid_len * sizeof(int));

    int threadsPerBlock;
    int blocksPerGrid;

    for(int i:Gpu_Db.DB_item_set){
        threadsPerBlock=Gpu_Db.max_sequence_len;
        blocksPerGrid=Gpu_Db.sid_len;
//        for(int j=0;j<Gpu_Db.sid_len;j++){
//
//        }

        //這裡之後改成依照每個sqe長度跑thread
        PEUcounter<<<blocksPerGrid,threadsPerBlock>>>(i,d_item_project,d_iu_project,d_ru_project,d_tid_project
                                                      ,d_sequence_len,Gpu_Db.sid_len,d_PEU_seq,d_Utility_seq,d_project_point);

        int *h_PEU_seq=new int[Gpu_Db.sid_len];
        hipMemcpy(h_PEU_seq, d_PEU_seq, Gpu_Db.sid_len*sizeof(int), hipMemcpyDeviceToHost);

        int *h_Utility_seq=new int[Gpu_Db.sid_len];
        hipMemcpy(h_Utility_seq, d_Utility_seq, Gpu_Db.sid_len*sizeof(int), hipMemcpyDeviceToHost);
        
        //這裡可以用加總
        int PEU_count=0;
        for(int j=0;j<Gpu_Db.sid_len;j++){
            PEU_count+=h_PEU_seq[j];
            if(PEU_count>=threshold){
                break;
            }
            //cout<<h_PEU_seq[j]<<endl;
        }

        if(PEU_count<threshold){
            continue;
        }

        //cout<<i<<endl;

        hipMemset(d_PEU_seq, 0, Gpu_Db.sid_len * sizeof(int));
        //cout<<"*/*****\n";
    }

//    PEUcounter<<<blocksPerGrid,threadsPerBlock>>>(821024,d_item_project,d_iu_project,d_ru_project,d_tid_project,d_sequence_len,Gpu_Db.sid_len,d_PEU_seq);
//
//    int *h_Result=new int[Gpu_Db.sid_len];
//    cudaMemcpy(h_Result, d_PEU_seq, Gpu_Db.sid_len*sizeof(int), cudaMemcpyDeviceToHost);
//
//    for(int j=0;j<Gpu_Db.sid_len;j++){
//        cout<<h_Result[j]<<endl;
//    }


    // 獲取 GPU 的內存信息
    status = hipMemGetInfo(&freeMem, &totalMem);

    if (status == hipSuccess) {
        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
    } else {
        cerr << "無法獲取內存信息，錯誤碼: " << hipGetErrorString(status) << endl;
    }

    for(int i=0;i<Gpu_Db.sid_len;i++) {
        hipFree(h_item_project[i]);
        hipFree(h_iu_project[i]);
        hipFree(h_ru_project[i]);
        hipFree(h_tid_project[i]);
    }

    delete[] h_item_project;
    delete[] h_iu_project;
    delete[] h_ru_project;
    delete[] h_tid_project;
    //delete[] h_Result;

    hipFree(d_item_project);
    hipFree(d_iu_project);
    hipFree(d_ru_project);
    hipFree(d_tid_project);

    hipFree(d_sequence_len);
    hipFree(d_PEU_seq);

//    // 獲取 GPU 的內存信息
//    status = cudaMemGetInfo(&freeMem, &totalMem);
//
//    if (status == cudaSuccess) {
//        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
//        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
//    } else {
//        cerr << "無法獲取內存信息，錯誤碼: " << cudaGetErrorString(status) << endl;
//    }


//    for(int i=0;i<Gpu_Db.sid_len;i++) {
//        for (int j = 0; j < Gpu_Db.sequence_len[i]; j++) {
//            cout << Gpu_Db.item[i][j] << " ";
//            cout << Gpu_Db.iu[i][j] << " ";
//            cout << Gpu_Db.ru[i][j] << " ";
//            cout << Gpu_Db.tid[i][j] << "\n";
//        }
//        cout << Gpu_Db.sequence_len[i] << endl;
//    }
    auto end = std::chrono::high_resolution_clock::now();


    // 計算持續時間，並轉換為毫秒
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds" << std::endl;

    return 0;
}
