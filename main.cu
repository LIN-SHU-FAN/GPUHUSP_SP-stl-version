
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <regex>
#include <fstream>
#include <unordered_map>
#include <unordered_set>
#include <map>
#include <stack>
#include <chrono>

using namespace std;
class DB{
public:
    vector<vector<int>> item,tid;
    vector<vector<int>> iu,ru;
    vector<int> sequence_len;

    vector<int> SWUtility;

    int DButility=0;

    unordered_map<int,int> item_swu;
    unordered_set<int> DB_item_set;

    map<int,map<int,vector<int>>> single_item_chain;//item->sid->vector(實例)
};

class GPU_DB {
public:
    vector<int> DB_item_set;//index->item
    unordered_map<int,int> DB_item_set_hash;//item->index

    int **item,**tid;
    int **iu,**ru;

    int sid_len;
    int *sequence_len;
    int max_sequence_len;


    int ***single_item_chain;//item->sid->instance (item已經做hash所以從0開始對應index)
    int **chain_sid;//長度是c_item_len 寬度是c_sid_len（紀錄single item 存在在哪些sid）

    int c_item_len;
    int *c_sid_len;//長度是c_item_len
    int max_c_sid_len;//哪個item出現在sid中最多次
    int **c_seq_len;//長度是c_item_len 寬度是c_sid_len

};

class Tree_node{
public:
    string pattern;
    int *ProjectArr_first_position,*ProjectArr_len;
};


//int test_max_seq=0;

void parseData(ifstream &file,DB &DBdata) {


    regex numberRegex("(\\d+)\\[(\\d+)\\]");

    vector<int> item,tid;
    vector<int> iu,ru;

    string line;
    while (getline(file, line)) {
        istringstream iss(line);
        int Tid = 1;
        int seq_len=0;
        string token;
        while (iss >> token) {
            if (token == "-1") {
                Tid++;
            } else if (token == "-2") {
                Tid = 1;

                DBdata.sequence_len.push_back(seq_len);
                seq_len=0;
            } else if (token.find("SUtility:") != string::npos) {
                int sUtility = stoi(token.substr(token.find(":") + 1));
                DBdata.SWUtility.push_back(sUtility);
                DBdata.DButility+=sUtility;
                unordered_set<int> uniqueSet(item.begin(),item.end());
                for(int i:uniqueSet){
                    if(DBdata.item_swu.find(i)!=DBdata.item_swu.end()){
                        DBdata.item_swu[i]+=sUtility;
                    }else{
                        DBdata.item_swu[i]=sUtility;
                    }
                }

//                if(item.size()>test_max_seq){
//                    test_max_seq=item.size();
//                }
                DBdata.item.push_back(item);
                DBdata.iu.push_back(iu);
                DBdata.ru.push_back(ru);
                DBdata.tid.push_back(tid);

                item.clear();
                iu.clear();
                ru.clear();
                tid.clear();
            } else {
                smatch match;
                if (regex_match(token, match, numberRegex)) {
                    int firstValue = stoi(match[1]);
                    int secondValue = stod(match[2]);
                    item.push_back(firstValue);
                    iu.push_back(secondValue);
                    ru.push_back(0);
                    tid.push_back(Tid);
                    seq_len++;
                }
            }
        }
    }

    int sid_len = int(DBdata.sequence_len.size());
    int ru_tmp;
    for(int i=0;i<sid_len;i++) {
        ru_tmp =0;
        for (int j = 0; j < DBdata.sequence_len[i]; j++) {
            ru_tmp +=DBdata.iu[i][j];
            DBdata.ru[i][j] =  DBdata.SWUtility[i] - ru_tmp;
        }
    }
}

void SWUpruning(double threshold,DB &DBdata){
    DB update_DB;
    vector<int> item,tid;
    vector<int> iu,ru;
    int seq_len;

    int sid_len = int(DBdata.sequence_len.size());
    for(int i=0;i<sid_len;i++){
        seq_len=0;
        for(int j=0;j<DBdata.sequence_len[i];j++){
            if(DBdata.item_swu[DBdata.item[i][j]]<threshold){
                for(int k=0;k<seq_len;k++){
                    ru[k]-=DBdata.iu[i][j];
                }
            }else{
                item.push_back(DBdata.item[i][j]);
                tid.push_back(DBdata.tid[i][j]);
                iu.push_back(DBdata.iu[i][j]);
                ru.push_back(DBdata.ru[i][j]);
                seq_len++;

                update_DB.DB_item_set.insert(DBdata.item[i][j]);
                update_DB.single_item_chain[DBdata.item[i][j]][update_DB.sequence_len.size()].push_back(item.size()-1);
            }
        }


        if(!item.empty()){
            update_DB.sequence_len.push_back(seq_len);

            update_DB.item.push_back(item);
            update_DB.iu.push_back(iu);
            update_DB.ru.push_back(ru);
            update_DB.tid.push_back(tid);

            item.clear();
            iu.clear();
            ru.clear();
            tid.clear();
        }

    }
    DBdata = update_DB;
}

void Bulid_GPU_DB(DB &DBdata,GPU_DB &Gpu_Db){
    //將item改成index表示並建立hash對應item->index
    Gpu_Db.DB_item_set.reserve(DBdata.DB_item_set.size());
    copy(DBdata.DB_item_set.begin(), DBdata.DB_item_set.end(),back_inserter(Gpu_Db.DB_item_set));
    sort(Gpu_Db.DB_item_set.begin(), Gpu_Db.DB_item_set.end());

    for (size_t i = 0; i < Gpu_Db.DB_item_set.size(); i++) {
        Gpu_Db.DB_item_set_hash[Gpu_Db.DB_item_set[i]] = static_cast<int>(i); // 值 -> 索引
    }

    Gpu_Db.sid_len=int(DBdata.sequence_len.size());
    Gpu_Db.sequence_len = new int[Gpu_Db.sid_len];

    Gpu_Db.item = new int*[Gpu_Db.sid_len];
    Gpu_Db.iu = new int*[Gpu_Db.sid_len];
    Gpu_Db.ru = new int*[Gpu_Db.sid_len];
    Gpu_Db.tid = new int*[Gpu_Db.sid_len];

    int max_sequence_len=0;

    for(int i=0;i<Gpu_Db.sid_len;i++){
        Gpu_Db.sequence_len[i] = DBdata.sequence_len[i];

        if(Gpu_Db.sequence_len[i]>max_sequence_len){
            max_sequence_len=Gpu_Db.sequence_len[i];
        }

        Gpu_Db.item[i]=DBdata.item[i].data();
        Gpu_Db.iu[i]=DBdata.iu[i].data();
        Gpu_Db.ru[i]=DBdata.ru[i].data();
        Gpu_Db.tid[i]=DBdata.tid[i].data();


//        Gpu_Db.item[i] = new int[Gpu_Db.sequence_len[i]];
//        Gpu_Db.iu[i] = new int[Gpu_Db.sequence_len[i]];
//        Gpu_Db.ru[i] = new int[Gpu_Db.sequence_len[i]];
//        Gpu_Db.tid[i] = new int[Gpu_Db.sequence_len[i]];



        for(int j=0;j<Gpu_Db.sequence_len[i];j++){
            Gpu_Db.item[i][j]=Gpu_Db.DB_item_set_hash[DBdata.item[i][j]];
//            Gpu_Db.iu[i][j]=DBdata.iu[i][j];
//            Gpu_Db.ru[i][j]=DBdata.ru[i][j];
//            Gpu_Db.tid[i][j]=DBdata.tid[i][j];

//            cout<<Gpu_Db.item[i][j]<<" ";
//            cout<<Gpu_Db.iu[i][j]<<" ";
//            cout<<Gpu_Db.ru[i][j]<<" ";
//            cout<<Gpu_Db.tid[i][j]<<"\n";
        }
//        cout<<"\n";

//        cout<<Gpu_Db.sequence_len[i]<<endl;

    }

    Gpu_Db.max_sequence_len=max_sequence_len;

    //Gpu_Db.DB_item_set=DBdata.DB_item_set;


    int item_len=DBdata.single_item_chain.size();

    Gpu_Db.single_item_chain = new int**[item_len];
    Gpu_Db.chain_sid = new int*[item_len]; //每個item出現在哪些sid

    Gpu_Db.c_item_len = item_len;
    Gpu_Db.c_sid_len = new int[item_len]; //每個item的sid長度
    Gpu_Db.c_seq_len = new int*[item_len]; //每個item在不同sid中的seq長度(實例)

    int i_index=0, sid_len, j_index=0,seq_len;
    int max_sid_len=0;
    for(auto i=DBdata.single_item_chain.begin();i!=DBdata.single_item_chain.end();i++){
        sid_len = i->second.size();
        Gpu_Db.single_item_chain[i_index] = new int*[sid_len];
        Gpu_Db.chain_sid[i_index] = new int[sid_len];

        Gpu_Db.c_sid_len[i_index] = sid_len;
        if(max_sid_len<sid_len){
            max_sid_len=sid_len;
        }
        Gpu_Db.c_seq_len[i_index] = new int[sid_len];

        j_index = 0;
        for(auto j = i->second.begin();j!=i->second.end();j++){
            seq_len = j->second.size();
            Gpu_Db.single_item_chain[i_index][j_index] = j->second.data();
            Gpu_Db.chain_sid[i_index][j_index] = j->first;

            Gpu_Db.c_seq_len[i_index][j_index] = seq_len;

            j_index++;
        }
        i_index++;
    }

    Gpu_Db.max_c_sid_len = max_sid_len;

}


__global__ void PEUcounter(int project_item,
                           int** d_item_project, int** d_iu_project, int** d_ru_project, int** d_tid_project,
                           int* d_sequence_len,int sid_len,
                           int *d_PEU_seq,int *d_Utility_seq) {

    if(threadIdx.x<d_sequence_len[blockIdx.x]){
        if(d_item_project[blockIdx.x][threadIdx.x]==project_item){
            //這裡應該能用redution在加速一點
            atomicMax(&d_PEU_seq[blockIdx.x], d_iu_project[blockIdx.x][threadIdx.x]+d_ru_project[blockIdx.x][threadIdx.x]);
            atomicMax(&d_Utility_seq[blockIdx.x], d_iu_project[blockIdx.x][threadIdx.x]);
        }
    }
}

__global__ void PeuCounter(int project_item,int sid,
                           int** d_item_project, int** d_iu_project, int** d_ru_project, int** d_tid_project,
                           int* d_sequence_len,int sid_len,
                           int *d_PEU_seq,int *d_Utility_seq,int *d_project_point) {
    if(threadIdx.x<d_sequence_len[sid]){
        if(d_item_project[sid][threadIdx.x]==project_item){
            atomicMax(&d_PEU_seq[sid], d_iu_project[sid][threadIdx.x]+d_ru_project[sid][threadIdx.x]);

        }
    }
}

__global__ void Array_add_reduction(int Array_len,int *inputArr,int *outputArr){
    __shared__ int shared_data[1024];
    int tid = threadIdx.x;
    /*
     * 假設有10個block，每個block有1024個thread
     * blockIdx.x = 0~9
     * blockDim.x = 1024
     * threadIdx.x = 0~1023
     * */
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    shared_data[tid] = (idx < Array_len) ? inputArr[idx] : 0;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        outputArr[blockIdx.x] = shared_data[0];
    }
}

__global__ void test(int ***d_single_item_chain,int d_c_item_len,int *d_c_sid_len,int **d_c_seq_len,int **d_chain_sid){
    for(int i=0;i<d_c_item_len;i++){
        printf("item:%d\n",i);
        for(int j=0;j<d_c_sid_len[i];j++){
            printf("sid:%d\n",d_chain_sid[i][j]);
            for(int k=0;k<d_c_seq_len[i][j];k++){
                printf("%d ",d_single_item_chain[i][j][k]);
            }
            printf("\n");
        }
    }
}

__global__ void test1(int *d_item,int *d_tid,int *d_iu,int *d_ru,int *d_offsets,int *d_sequence_len,int d_sid_len){
    for(int i = 0;i<d_sid_len;i++){
        for(int j=d_offsets[i];j<d_offsets[i]+d_sequence_len[i];j++){
            printf("i[%d]j[%d] = %d ",i,j-d_offsets[i],d_item[j]);
        }

        printf("sequence_len:%d\n",d_sequence_len[i]);
    }
    printf("%d\n",d_sid_len);
}



__global__ void test2(int *d_flat_single_item_chain,int *d_chain_offsets_level1,int *d_chain_offsets_level2,
                      int *d_flat_chain_sid,int *d_chain_sid_offsets,
                      int d_c_item_len,
                      int *d_c_sid_len,
                      int *d_flat_c_seq_len,int *d_c_seq_len_offsets
                      ){

    //index = offsets_level2[offsets_level1[item] + sid] + instance 三維陣列

    //int index = d_c_seq_len_offsets[item] + sid; 二維陣列
    //int value = d_flat_c_seq_len[index];

    for(int i = 0;i<d_c_item_len;i++){
        for(int j=0;j<d_c_sid_len[i];j++){
            for(int k=0;k<d_flat_c_seq_len[d_c_seq_len_offsets[i]+j];k++){
                printf("item[%d]sid[%d]instance[%d] = %d \n",i,d_flat_chain_sid[d_chain_sid_offsets[i]+j],k,d_flat_single_item_chain[d_chain_offsets_level2[d_chain_offsets_level1[i]+j]+k]);
            }
        }
    }
//    for(int i = 0;i<d_sid_len;i++){
//        for(int j=d_offsets[i];j<d_offsets[i]+d_sequence_len[i];j++){
//            printf("i[%d]j[%d] = %d ",i,j-d_offsets[i],d_item[j]);
//        }
//
//        printf("sequence_len:%d\n",d_sequence_len[i]);
//    }
//    printf("%d\n",d_sid_len);
}

__global__ void count_chain_memory_size(int d_c_item_len,
                                        int *d_c_sid_len,
                                        int *d_flat_c_seq_len,int *d_c_seq_len_offsets,
                                        int *d_flat_single_item_chain,int *d_chain_offsets_level1,int *d_chain_offsets_level2,
                                        int *d_item_memory_overall_size){
    
    
    __shared__ int sub_data[1024];//把資料寫到shared memory且縮小到1024內（如果有超過1024）且順便用梯形公式算好

    //index = d_chain_offsets_level2[d_chain_offsets_level1[item] + sid] + instance 三維陣列

    //int index = d_c_seq_len_offsets[item] + sid; 二維陣列
    //int value = d_flat_c_seq_len[index];
    
    int sum = 0,first_project,seq_len,n;
    //d_c_sid_len[blockIdx.x]=>每個item的sid數量
    for (int i = threadIdx.x; i < d_c_sid_len[blockIdx.x]; i += blockDim.x) {
        first_project = d_flat_single_item_chain[d_chain_offsets_level2[d_chain_offsets_level1[blockIdx.x] + i] + 0];//blockIdx.x對應item,i對應sid
        seq_len = d_flat_c_seq_len[d_c_seq_len_offsets[blockIdx.x] + i ];
        n = seq_len - first_project - 1;
        sum += (n+1)*n/2;//梯形公式
    }
    
    sub_data[threadIdx.x] = sum;
    __syncthreads();

    // 使用平行 reduction 計算陣列的總和
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (blockDim.x < s) {
            sub_data[blockDim.x] += sub_data[blockDim.x + s];
        }
        __syncthreads();
    }
    
    
}


void GPUHUSP(const GPU_DB &Gpu_Db){
    vector<int> flat_item, flat_tid, flat_iu, flat_ru;
    vector<int> db_offsets;
    int offset = 0;
    //project DB初始
    for (int i = 0; i < Gpu_Db.sid_len; i++) {
        db_offsets.push_back(offset);
        for (int j = 0; j < Gpu_Db.sequence_len[i]; j++) {
            flat_item.push_back(Gpu_Db.item[i][j]);
            flat_tid.push_back(Gpu_Db.tid[i][j]);
            flat_iu.push_back(Gpu_Db.iu[i][j]);
            flat_ru.push_back(Gpu_Db.ru[i][j]);
        }
        offset += Gpu_Db.sequence_len[i];
    }

    int *d_item, *d_tid, *d_iu, *d_ru;
    int *d_db_offsets, *d_sequence_len;//offsets裡面存陣列偏移量 從0開始

    hipMalloc(&d_item, flat_item.size() * sizeof(int));
    hipMalloc(&d_tid, flat_tid.size() * sizeof(int));
    hipMalloc(&d_iu, flat_iu.size() * sizeof(int));
    hipMalloc(&d_ru, flat_ru.size() * sizeof(int));
    hipMalloc(&d_db_offsets, db_offsets.size() * sizeof(int));
    hipMalloc(&d_sequence_len, Gpu_Db.sid_len * sizeof(int));

    hipMemcpy(d_item, flat_item.data(), flat_item.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_tid, flat_tid.data(), flat_tid.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_iu, flat_iu.data(), flat_iu.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ru, flat_ru.data(), flat_ru.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_db_offsets, db_offsets.data(), db_offsets.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sequence_len, Gpu_Db.sequence_len, Gpu_Db.sid_len * sizeof(int), hipMemcpyHostToDevice);

//    test1<<<1,1>>>(d_item,d_tid,d_iu,d_ru,d_db_offsets,d_sequence_len,Gpu_Db.sid_len);
//    cudaDeviceSynchronize();
//    cout<<"\n";

    /*
     * 假設 single_item_chain 為以下結構：
        Item 0:
          SID 0: [1, 2, 3]
          SID 1: [4, 5]

        Item 1:
          SID 0: [6]
          SID 1: [7, 8, 9, 10]
          SID 2: [11, 12]

        Item 2:
          SID 0: [13, 14, 15, 16, 17]


        offsets_level1=[0, 2, 5, 6]
        對於 Item 0，二維陣列在 flat_chain 的偏移量為 0。
        對於 Item 1，二維陣列在 flat_chain 的偏移量為 2（Item 0 有 2 個 sid）。
        對於 Item 2，二維陣列在 flat_chain 的偏移量為 5（Item 0 和 Item 1 共計 5 個 sid）。

        offsets_level2=[0, 3, 5, 6, 10, 12, 13]
        對於 Item 0, SID 0，偏移量為 0。
        對於 Item 0, SID 1，偏移量為 3。
        對於 Item 1, SID 0，偏移量為 5。
        ...


        offsets_level1：記錄每個 item 在 offsets_level2 中的 "起始" 索引。
        offsets_level2：記錄每個 sid 在 flat_single_item_chain 中的 "起始" 索引。

        三維陣列
        single_item_chain[item][sid][instance]
        等於
        index = offsets_level2[offsets_level1[item] + sid] + instance

        二維陣列
        int index = d_c_seq_len_offsets[row] + col;
        int value = d_flat_c_seq_len[index];
    */
    //single item chain 初始
    vector<int> flat_single_item_chain;
    vector<int> chain_offsets_level1(Gpu_Db.c_item_len + 1, 0);// 長度為 c_item_len + 1，最後一個值是總 sid 數量
    vector<int> chain_offsets_level2;

    vector<int> flat_chain_sid;
    vector<int> chain_sid_offsets;
    int chain_sid_offset=0;

    vector<int> flat_c_seq_len;
    vector<int> c_seq_len_offsets;
    int c_seq_len_offset=0;

    for(int i=0;i<Gpu_Db.c_item_len;i++){
        chain_sid_offsets.push_back(chain_sid_offset);

        c_seq_len_offsets.push_back(c_seq_len_offset);
        for(int j=0;j<Gpu_Db.c_sid_len[i];j++){
            // 將偏移量存入 chain_offsets_level2
            chain_offsets_level2.push_back(int(flat_single_item_chain.size()));

            for(int k=0;k<Gpu_Db.c_seq_len[i][j];k++){
                flat_single_item_chain.push_back(Gpu_Db.single_item_chain[i][j][k]);
            }

            flat_chain_sid.push_back(Gpu_Db.chain_sid[i][j]);

            flat_c_seq_len.push_back(Gpu_Db.c_seq_len[i][j]);
        }

        // 計算 offsets_level1_chain
        chain_offsets_level1[i + 1] = int(chain_offsets_level2.size());

        chain_sid_offset+=Gpu_Db.c_sid_len[i];

        c_seq_len_offset+=Gpu_Db.c_sid_len[i];
    }

    int *d_flat_single_item_chain,*d_chain_offsets_level1,*d_chain_offsets_level2;

    hipMalloc(&d_flat_single_item_chain, flat_single_item_chain.size() * sizeof(int));
    hipMalloc(&d_chain_offsets_level1, chain_offsets_level1.size() * sizeof(int));
    hipMalloc(&d_chain_offsets_level2, chain_offsets_level2.size() * sizeof(int));

    hipMemcpy(d_flat_single_item_chain, flat_single_item_chain.data(), flat_single_item_chain.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_chain_offsets_level1, chain_offsets_level1.data(), chain_offsets_level1.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_chain_offsets_level2, chain_offsets_level2.data(), chain_offsets_level2.size() * sizeof(int), hipMemcpyHostToDevice);

    int *d_flat_chain_sid,*d_chain_sid_offsets;
    hipMalloc(&d_flat_chain_sid, flat_chain_sid.size() * sizeof(int));
    hipMalloc(&d_chain_sid_offsets, chain_sid_offsets.size() * sizeof(int));

    hipMemcpy(d_flat_chain_sid, flat_chain_sid.data(), flat_chain_sid.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_chain_sid_offsets, chain_sid_offsets.data(), chain_sid_offsets.size() * sizeof(int), hipMemcpyHostToDevice);

    int *d_flat_c_seq_len,*d_c_seq_len_offsets;
    hipMalloc(&d_flat_c_seq_len, flat_c_seq_len.size() * sizeof(int));
    hipMalloc(&d_c_seq_len_offsets, c_seq_len_offsets.size() * sizeof(int));

    hipMemcpy(d_flat_c_seq_len, flat_c_seq_len.data(), flat_c_seq_len.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c_seq_len_offsets, c_seq_len_offsets.data(), c_seq_len_offsets.size() * sizeof(int), hipMemcpyHostToDevice);

    int *d_c_sid_len;
    hipMalloc(&d_c_sid_len, Gpu_Db.c_item_len * sizeof(int));
    hipMemcpy(d_c_sid_len, Gpu_Db.c_sid_len , Gpu_Db.c_item_len * sizeof(int), hipMemcpyHostToDevice);


//    test2<<<1,1>>>(d_flat_single_item_chain,d_chain_offsets_level1,d_chain_offsets_level2,
//                   d_flat_chain_sid,d_chain_sid_offsets,
//                   Gpu_Db.c_item_len,
//                   d_c_sid_len,
//                   d_flat_c_seq_len,d_c_seq_len_offsets
//                   );
//    cudaDeviceSynchronize();
//    cout<<"";


    //算DFS所需要的最大空間（用最壞情況估 a,aa,a...）    

    int *d_item_memory_overall_size;
    //每個item在每個投影seq中第一個投影點到投影資料庫的末端長度算梯形公式的加總
    //長度是c_item_len

    hipMalloc(&d_item_memory_overall_size,Gpu_Db.c_item_len * sizeof(int));
    hipMemset(d_item_memory_overall_size, 0, Gpu_Db.c_item_len * sizeof(int));

    int thread_num=0;
    if(Gpu_Db.max_c_sid_len>1024){
        thread_num=1024;
    }else{
        thread_num=Gpu_Db.max_c_sid_len;
    }
    
    count_chain_memory_size<<<Gpu_Db.c_item_len,thread_num>>>(Gpu_Db.c_item_len,
                                                        d_c_sid_len,
                                                        d_flat_chain_sid,d_chain_sid_offsets,
                                                        d_flat_single_item_chain,d_chain_offsets_level1,d_chain_offsets_level2,
                                                        d_item_memory_overall_size);

//    size_t freeMem = 0;
//    size_t totalMem = 0;
//
//    // 獲取 GPU 的內存信息
//    cudaError_t status = cudaMemGetInfo(&freeMem, &totalMem);
//
//    if (status == cudaSuccess) {
//        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
//        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
//    } else {
//        cerr << "無法獲取內存信息，錯誤碼: " << cudaGetErrorString(status) << endl;
//    }
//
//    //project DB初始
//    int **d_item_project;
//    cudaMalloc(&d_item_project, Gpu_Db.sid_len * sizeof(int*));
//
//    int **d_iu_project;
//    cudaMalloc(&d_iu_project, Gpu_Db.sid_len * sizeof(int*));
//
//    int **d_ru_project;
//    cudaMalloc(&d_ru_project, Gpu_Db.sid_len * sizeof(int*));
//
//    int **d_tid_project;
//    cudaMalloc(&d_tid_project, Gpu_Db.sid_len * sizeof(int*));
//
//    // 主機上的指標陣列，用於存放每一行的 d_tmp 指標
//    //cudaMemcpy(&d_item_project[i], &d_tmp, sizeof(int*), cudaMemcpyDeviceToDevice);不能這樣,因為d_item_project[i]在host不能讀取
//    //要開主機指標的指標（陣列）存Device指標
//    int **h_item_project = new int*[Gpu_Db.sid_len];
//    int **h_iu_project = new int*[Gpu_Db.sid_len];
//    int **h_ru_project = new int*[Gpu_Db.sid_len];
//    int **h_tid_project = new int*[Gpu_Db.sid_len];
//
//
//    for(int i=0;i<Gpu_Db.sid_len;i++){
//        int *d_tmp;
//        cudaMalloc(&d_tmp, Gpu_Db.sequence_len[i] * sizeof(int));
//        cudaMemcpy(d_tmp,Gpu_Db.item[i],Gpu_Db.sequence_len[i]*sizeof(int),cudaMemcpyHostToDevice);
//        h_item_project[i] = d_tmp;
//
//        cudaMalloc(&d_tmp, Gpu_Db.sequence_len[i] * sizeof(int));
//        cudaMemcpy(d_tmp,Gpu_Db.iu[i],Gpu_Db.sequence_len[i]*sizeof(int),cudaMemcpyHostToDevice);
//        h_iu_project[i] = d_tmp;
//
//        cudaMalloc(&d_tmp, Gpu_Db.sequence_len[i] * sizeof(int));
//        cudaMemcpy(d_tmp,Gpu_Db.ru[i],Gpu_Db.sequence_len[i]*sizeof(int),cudaMemcpyHostToDevice);
//        h_ru_project[i] = d_tmp;
//
//        cudaMalloc(&d_tmp, Gpu_Db.sequence_len[i] * sizeof(int));
//        cudaMemcpy(d_tmp,Gpu_Db.tid[i],Gpu_Db.sequence_len[i]*sizeof(int),cudaMemcpyHostToDevice);
//        h_tid_project[i] = d_tmp;
//    }
//
//    cudaMemcpy(d_item_project, h_item_project, Gpu_Db.sid_len * sizeof(int*), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_iu_project, h_iu_project, Gpu_Db.sid_len * sizeof(int*), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_ru_project, h_ru_project, Gpu_Db.sid_len * sizeof(int*), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_tid_project, h_tid_project, Gpu_Db.sid_len * sizeof(int*), cudaMemcpyHostToDevice);
//
//
//    int *d_sequence_len;
//    cudaMalloc(&d_sequence_len, Gpu_Db.sid_len * sizeof(int));
//    cudaMemcpy(d_sequence_len,Gpu_Db.sequence_len,Gpu_Db.sid_len * sizeof(int),cudaMemcpyHostToDevice);
//
//
//    //single item chain 初始
//    int ***d_single_item_chain;
//    cudaMalloc((void ***)&d_single_item_chain, Gpu_Db.c_item_len * sizeof(int **));
//
//    int ***h_three_dimension_single_item_chain = new int**[Gpu_Db.c_item_len];//用來存device指標的三維指標
//    int **h_two_dimension_single_item_chain;
//
//    int ***h_delete_tmp = new int**[Gpu_Db.c_item_len];//用來cudafree用的
//
//    for(int i=0;i<Gpu_Db.c_item_len;i++){
//        h_two_dimension_single_item_chain = new int*[Gpu_Db.c_sid_len[i]];//用來存device指標的二維指標
//        for(int j=0;j<Gpu_Db.c_sid_len[i];j++){
//            int *d_one_dimension_tmp;
//            cudaMalloc(&d_one_dimension_tmp,Gpu_Db.c_seq_len[i][j]*sizeof(int));
//            cudaMemcpy(d_one_dimension_tmp,Gpu_Db.single_item_chain[i][j],Gpu_Db.c_seq_len[i][j]*sizeof(int),cudaMemcpyHostToDevice);
//            h_two_dimension_single_item_chain[j] = d_one_dimension_tmp;
//        }
//
//        int **d_two_dimension_tmp;
//        cudaMalloc(&d_two_dimension_tmp,Gpu_Db.c_sid_len[i]*sizeof(int*));
//        cudaMemcpy(d_two_dimension_tmp,h_two_dimension_single_item_chain,Gpu_Db.c_sid_len[i]*sizeof(int*),cudaMemcpyHostToDevice);
//
//        h_three_dimension_single_item_chain[i] = d_two_dimension_tmp;
//        h_delete_tmp[i]=h_two_dimension_single_item_chain;
//
//        //delete[] h_two_dimension_single_item_chain;
//    }
//    cudaMemcpy(d_single_item_chain,h_three_dimension_single_item_chain,Gpu_Db.c_item_len*sizeof(int**),cudaMemcpyHostToDevice);
//    //delete[] h_three_dimension_single_item_chain;
//
//    int *d_c_sid_len;
//    cudaMalloc(&d_c_sid_len,Gpu_Db.c_item_len*sizeof(int));
//    cudaMemcpy(d_c_sid_len,Gpu_Db.c_sid_len,Gpu_Db.c_item_len*sizeof(int),cudaMemcpyHostToDevice);
//
//    int **d_c_seq_len;
//    cudaMalloc(&d_c_seq_len,Gpu_Db.c_item_len*sizeof(int*));
//    int **h_tmp_c_seq_len = new int*[Gpu_Db.c_item_len];
//
//    int **d_chain_sid;
//    cudaMalloc(&d_chain_sid,Gpu_Db.c_item_len*sizeof(int*));
//    int **h_tmp_chain_sid = new int*[Gpu_Db.c_item_len];
//
//    for(int i=0;i<Gpu_Db.c_item_len;i++){
//        int *d_tmp;
//        cudaMalloc(&d_tmp,Gpu_Db.c_sid_len[i]*sizeof(int));
//        cudaMemcpy(d_tmp,Gpu_Db.c_seq_len[i],Gpu_Db.c_sid_len[i]*sizeof(int),cudaMemcpyHostToDevice);
//        h_tmp_c_seq_len[i] = d_tmp;
//
//        cudaMalloc(&d_tmp,Gpu_Db.c_sid_len[i]*sizeof(int));
//        cudaMemcpy(d_tmp,Gpu_Db.chain_sid[i],Gpu_Db.c_sid_len[i]*sizeof(int),cudaMemcpyHostToDevice);
//        h_tmp_chain_sid[i] = d_tmp;
//    }
//
//    cudaMemcpy(d_c_seq_len,h_tmp_c_seq_len,Gpu_Db.c_item_len*sizeof(int*),cudaMemcpyHostToDevice);
//
//    cudaMemcpy(d_chain_sid,h_tmp_chain_sid,Gpu_Db.c_item_len*sizeof(int*),cudaMemcpyHostToDevice);
//
//
//
//    int *d_PEU_Arr;//長度是seq數量 用來算peu(可重用)
//    cudaMalloc(&d_PEU_Arr,Gpu_Db.sid_len*sizeof(int));
//    cudaMemset(d_PEU_Arr, 0, Gpu_Db.sid_len * sizeof(int));
//
//    int *d_I_list_RSU_Arr,*d_S_list_RSU_Arr;//長度是item數量 用來算rsu(可重用)
//    cudaMalloc(&d_I_list_RSU_Arr,Gpu_Db.c_item_len*sizeof(int));
//    cudaMalloc(&d_S_list_RSU_Arr,Gpu_Db.c_item_len*sizeof(int));
//    cudaMemset(d_I_list_RSU_Arr, 0, Gpu_Db.c_item_len * sizeof(int));
//    cudaMemset(d_S_list_RSU_Arr, 0, Gpu_Db.c_item_len * sizeof(int));
//
//
//
//
//    test<<<1,1>>>(d_single_item_chain,Gpu_Db.c_item_len,d_c_sid_len,d_c_seq_len,d_chain_sid);
//
////    for(int i=0;i<Gpu_Db.sid_len;i++){
////        for(int j=0;j<Gpu_Db.sequence_len[i];j++){
////            cout<<Gpu_Db.item[i][j]<<" ";
////        }
////        //cout<<Gpu_Db.sequence_len[i]<<endl;
////        cout<<endl;
////    }
//
//    stack<Tree_node> Main_stack;
//    Tree_node top1_Node;
//
//    top1_Node.pattern = "";
//    top1_Node.ProjectArr_first_position=new int[Gpu_Db.sid_len];
//    top1_Node.ProjectArr_len=new int[Gpu_Db.sid_len];
//
//    //DB clearDBdata;
//    //DBdata = clearDBdata;
//    //cout<<Gpu_Db.max_sequence_len;
//
//
//
//
//
//
//
//
//
//
////    for(int i=0;i<Gpu_Db.sid_len;i++){
////        cout<<Gpu_Db.sequence_len[i]<<endl;
////    }
////    cout<<endl;
//    //kernelfunction操作
//
//    //如果sequence長度有超過1024需要額外處理
//    //int *block_per_sequence = new int[Gpu_Db.sid_len];
////    if(Gpu_Db.max_sequence_len>1024){
////        threadsPerBlock=1024;
////        for(int i=0;i<Gpu_Db.sid_len;i++){
////            blocksPerGrid+= (Gpu_Db.sequence_len[i]+threadsPerBlock-1)/threadsPerBlock;
////            block_per_sequence[i] = (Gpu_Db.sequence_len[i]+threadsPerBlock-1)/threadsPerBlock;
////        }
////    }else{
////        threadsPerBlock=Gpu_Db.max_sequence_len;
////        blocksPerGrid=Gpu_Db.sid_len;
////    }
//
//    int *d_PEU_seq;
//    cudaMalloc(&d_PEU_seq, Gpu_Db.sid_len * sizeof(int));
//    cudaMemset(d_PEU_seq, 0, Gpu_Db.sid_len * sizeof(int));
//
//    int d_PEU_add_len = (Gpu_Db.sid_len + 1024 - 1) / 1024;//看有多少seq在用1024切
//    int *d_PEU_add_result;
//    cudaMalloc(&d_PEU_add_result, d_PEU_add_len * sizeof(int));
//
//    int *d_Utility_seq;
//    cudaMalloc(&d_Utility_seq, Gpu_Db.sid_len * sizeof(int));
//    cudaMemset(d_Utility_seq, 0, Gpu_Db.sid_len * sizeof(int));
//
//
//
//    int threadsPerBlock;
//    int blocksPerGrid;
//
//
//
//    for(int i:Gpu_Db.DB_item_set){
//        threadsPerBlock=Gpu_Db.max_sequence_len;
//        blocksPerGrid=Gpu_Db.sid_len;
////        for(int j=0;j<Gpu_Db.sid_len;j++){
////            PeuCounter<<<1,Gpu_Db.sequence_len[j]>>>(i,j,d_item_project,d_iu_project,d_ru_project,d_tid_project
////                    ,d_sequence_len,Gpu_Db.sid_len,d_PEU_seq,d_Utility_seq,d_project_point);
////        }
//
//        //一次開好比較快
//        PEUcounter<<<blocksPerGrid,threadsPerBlock>>>(i,d_item_project,d_iu_project,d_ru_project,d_tid_project
//                ,d_sequence_len,Gpu_Db.sid_len,d_PEU_seq,d_Utility_seq);
//
////        int *h_PEU_seq=new int[Gpu_Db.sid_len];
////        cudaMemcpy(h_PEU_seq, d_PEU_seq, Gpu_Db.sid_len*sizeof(int), cudaMemcpyDeviceToHost);
////
////        //這裡可以用加總
////        int PEU_count=0;
////        for(int j=0;j<Gpu_Db.sid_len;j++){
////            PEU_count+=h_PEU_seq[j];
////        }
////        cout<<"item:"<<i<<endl;
////        cout<<"PEU:"<<PEU_count<<endl<<endl;
//        //
////        int *h_Utility_seq=new int[Gpu_Db.sid_len];
////        cudaMemcpy(h_Utility_seq, d_Utility_seq, Gpu_Db.sid_len*sizeof(int), cudaMemcpyDeviceToHost);
//
//
//        Array_add_reduction<<<d_PEU_add_len,1024>>>(Gpu_Db.sid_len,d_PEU_seq,d_PEU_add_result);
//        int *h_PEU_add_result=new int[d_PEU_add_len];
//        cudaMemcpy(h_PEU_add_result, d_PEU_add_result, d_PEU_add_len*sizeof(int), cudaMemcpyDeviceToHost);
//
//        int PEU_count = 0;
//        for (int j = 0; j < d_PEU_add_len; j++) {
//            PEU_count += h_PEU_add_result[j];
//        }
////        cout<<"item:"<<i<<endl;
////        cout<<"PEU:"<<PEU_count<<endl;
//
//
//
//
//
////        if(PEU_count<threshold){
////            continue;
////        }
//
//        cudaMemset(d_PEU_seq, 0, Gpu_Db.sid_len * sizeof(int));
//        //cout<<"*/*****\n";
//    }
//
//
//
////    PEUcounter<<<blocksPerGrid,threadsPerBlock>>>(821024,d_item_project,d_iu_project,d_ru_project,d_tid_project,d_sequence_len,Gpu_Db.sid_len,d_PEU_seq);
////
////    int *h_Result=new int[Gpu_Db.sid_len];
////    cudaMemcpy(h_Result, d_PEU_seq, Gpu_Db.sid_len*sizeof(int), cudaMemcpyDeviceToHost);
////
////    for(int j=0;j<Gpu_Db.sid_len;j++){
////        cout<<h_Result[j]<<endl;
////    }
//
//    //project free
//    cudaFree(d_item_project);
//    cudaFree(d_iu_project);
//    cudaFree(d_ru_project);
//    cudaFree(d_tid_project);
//
//    for(int i=0;i<Gpu_Db.sid_len;i++) {
//        cudaFree(h_item_project[i]);
//        cudaFree(h_iu_project[i]);
//        cudaFree(h_ru_project[i]);
//        cudaFree(h_tid_project[i]);
//    }
//
//    delete[] h_item_project;
//    delete[] h_iu_project;
//    delete[] h_ru_project;
//    delete[] h_tid_project;
//    //delete[] h_Result;
//
//    //chain free
//    cudaFree(d_single_item_chain);
//
//    for(int i=0;i<Gpu_Db.c_item_len;i++){
//        cudaFree(h_three_dimension_single_item_chain[i]);
//        for(int j=0;j<Gpu_Db.c_sid_len[i];j++){
//            cudaFree(h_delete_tmp[i][j]);
//        }
//    }
//
//    delete[] h_three_dimension_single_item_chain;
//    delete[] h_delete_tmp;
//
//    cudaFree(d_c_seq_len);
//    cudaFree(d_chain_sid);
//
//    for(int i=0;i<Gpu_Db.c_item_len;i++){
//        cudaFree(h_tmp_c_seq_len[i]);
//        cudaFree(h_tmp_chain_sid[i]);
//    }
//
//    delete [] h_tmp_c_seq_len;
//    delete [] h_tmp_chain_sid;
//
//
//    cudaFree(d_sequence_len);
//    cudaFree(d_PEU_seq);
//
//    // 獲取 GPU 的內存信息
//    status = cudaMemGetInfo(&freeMem, &totalMem);
//
//    if (status == cudaSuccess) {
//        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
//        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
//    } else {
//        cerr << "無法獲取內存信息，錯誤碼: " << cudaGetErrorString(status) << endl;
//    }
////    // 獲取 GPU 的內存信息
////    status = cudaMemGetInfo(&freeMem, &totalMem);
////
////    if (status == cudaSuccess) {
////        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
////        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
////    } else {
////        cerr << "無法獲取內存信息，錯誤碼: " << cudaGetErrorString(status) << endl;
////    }
//
//
////    for(int i=0;i<Gpu_Db.sid_len;i++) {
////        for (int j = 0; j < Gpu_Db.sequence_len[i]; j++) {
////            cout << Gpu_Db.item[i][j] << " ";
////            cout << Gpu_Db.iu[i][j] << " ";
////            cout << Gpu_Db.ru[i][j] << " ";
////            cout << Gpu_Db.tid[i][j] << "\n";
////        }
////        cout << Gpu_Db.sequence_len[i] << endl;
////    }

}


int main() {
    // 指定要讀取的檔案名稱
    string filename = "YoochooseSamller.txt";
    //string filename = "Yoochoose.txt";
    ifstream file(filename);
    vector<string> lines;

    // 檢查檔案是否成功開啟
    if (!file.is_open()) {
        cerr << "無法開啟檔案: " << filename << endl;
        return 1; // 返回錯誤代碼
    }


    DB DBdata;
    parseData(file,DBdata);

//    cout<<test_max_seq;
    file.close(); // 關閉檔案

    double threshold = 0.01 * DBdata.DButility;
    //double threshold = 0.00024 * DBdata.DButility;


    SWUpruning(threshold,DBdata);


    GPU_DB Gpu_Db;


    Bulid_GPU_DB(DBdata,Gpu_Db);


//    for(int i=0;i<Gpu_Db.c_item_len;i++){
//        for(int j=0;j<Gpu_Db.c_sid_len[i];j++){
//            cout<<Gpu_Db.chain_sid[i][j]<<"\n";
//            for(int k=0;k<Gpu_Db.c_seq_len[i][j];k++){
//                cout<<Gpu_Db.single_item_chain[i][j][k]<<" "<<endl;
//            }
//        }
//    }



    auto start = std::chrono::high_resolution_clock::now();

    GPUHUSP(Gpu_Db);

    auto end = std::chrono::high_resolution_clock::now();
    // 計算持續時間，並轉換為毫秒
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds" << std::endl;



    return 0;
}
