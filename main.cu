
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <regex>
#include <fstream>
#include <unordered_map>
#include <unordered_set>
#include <map>
#include <stack>
#include <chrono>

using namespace std;
class DB{
public:
    vector<vector<int>> item,tid;
    vector<vector<int>> iu,ru;
    vector<int> sequence_len;

    vector<int> SWUtility;

    int DButility=0;

    unordered_map<int,int> item_swu;
    unordered_set<int> DB_item_set;

    map<int,map<int,vector<int>>> single_item_chain;//item->sid->vector(實例)
};

class GPU_DB {
public:
    vector<int> DB_item_set;//index->item
    unordered_map<int,int> DB_item_set_hash;//item->index

    int **item,**tid;
    int **iu,**ru;

    int sid_len;
    int *sequence_len;
    int max_sequence_len;


    int ***single_item_chain;//item->sid->instance (item已經做hash所以從0開始對應index)
    int **chain_sid;//長度是c_item_len 寬度是c_sid_len

    int c_item_len;
    int *c_sid_len;//長度是c_item_len
    int **c_seq_len;//長度是c_item_len 寬度是c_sid_len

};
class Tree_node{
public:
    string pattern;
    int *ProjectArr_first_position,*ProjectArr_len;
};


//int test_max_seq=0;

void parseData(ifstream &file,DB &DBdata) {


    regex numberRegex("(\\d+)\\[(\\d+)\\]");

    vector<int> item,tid;
    vector<int> iu,ru;

    string line;
    while (getline(file, line)) {
        istringstream iss(line);
        int Tid = 1;
        int seq_len=0;
        string token;
        while (iss >> token) {
            if (token == "-1") {
                Tid++;
            } else if (token == "-2") {
                Tid = 1;

                DBdata.sequence_len.push_back(seq_len);
                seq_len=0;
            } else if (token.find("SUtility:") != string::npos) {
                int sUtility = stoi(token.substr(token.find(":") + 1));
                DBdata.SWUtility.push_back(sUtility);
                DBdata.DButility+=sUtility;
                unordered_set<int> uniqueSet(item.begin(),item.end());
                for(int i:uniqueSet){
                    if(DBdata.item_swu.find(i)!=DBdata.item_swu.end()){
                        DBdata.item_swu[i]+=sUtility;
                    }else{
                        DBdata.item_swu[i]=sUtility;
                    }
                }

//                if(item.size()>test_max_seq){
//                    test_max_seq=item.size();
//                }
                DBdata.item.push_back(item);
                DBdata.iu.push_back(iu);
                DBdata.ru.push_back(ru);
                DBdata.tid.push_back(tid);

                item.clear();
                iu.clear();
                ru.clear();
                tid.clear();
            } else {
                smatch match;
                if (regex_match(token, match, numberRegex)) {
                    int firstValue = stoi(match[1]);
                    int secondValue = stod(match[2]);
                    item.push_back(firstValue);
                    iu.push_back(secondValue);
                    ru.push_back(0);
                    tid.push_back(Tid);
                    seq_len++;
                }
            }
        }
    }

    int sid_len = int(DBdata.sequence_len.size());
    int ru_tmp;
    for(int i=0;i<sid_len;i++) {
        ru_tmp =0;
        for (int j = 0; j < DBdata.sequence_len[i]; j++) {
            ru_tmp +=DBdata.iu[i][j];
            DBdata.ru[i][j] =  DBdata.SWUtility[i] - ru_tmp;
        }
    }
}

void SWUpruning(double threshold,DB &DBdata){
    DB update_DB;
    vector<int> item,tid;
    vector<int> iu,ru;
    int seq_len;

    int sid_len = int(DBdata.sequence_len.size());
    for(int i=0;i<sid_len;i++){
        seq_len=0;
        for(int j=0;j<DBdata.sequence_len[i];j++){
            if(DBdata.item_swu[DBdata.item[i][j]]<threshold){
                for(int k=0;k<seq_len;k++){
                    ru[k]-=DBdata.iu[i][j];
                }
            }else{
                item.push_back(DBdata.item[i][j]);
                tid.push_back(DBdata.tid[i][j]);
                iu.push_back(DBdata.iu[i][j]);
                ru.push_back(DBdata.ru[i][j]);
                seq_len++;

                update_DB.DB_item_set.insert(DBdata.item[i][j]);
                update_DB.single_item_chain[DBdata.item[i][j]][update_DB.sequence_len.size()].push_back(item.size()-1);
            }
        }


        if(!item.empty()){
            update_DB.sequence_len.push_back(seq_len);

            update_DB.item.push_back(item);
            update_DB.iu.push_back(iu);
            update_DB.ru.push_back(ru);
            update_DB.tid.push_back(tid);

            item.clear();
            iu.clear();
            ru.clear();
            tid.clear();
        }

    }
    DBdata = update_DB;
}

void Bulid_GPU_DB(DB &DBdata,GPU_DB &Gpu_Db){
    //將item改成index表示並建立hash對應item->index
    Gpu_Db.DB_item_set.reserve(DBdata.DB_item_set.size());
    copy(DBdata.DB_item_set.begin(), DBdata.DB_item_set.end(),back_inserter(Gpu_Db.DB_item_set));
    sort(Gpu_Db.DB_item_set.begin(), Gpu_Db.DB_item_set.end());

    for (size_t i = 0; i < Gpu_Db.DB_item_set.size(); i++) {
        Gpu_Db.DB_item_set_hash[Gpu_Db.DB_item_set[i]] = static_cast<int>(i); // 值 -> 索引
    }

    Gpu_Db.sid_len=int(DBdata.sequence_len.size());
    Gpu_Db.sequence_len = new int[Gpu_Db.sid_len];

    Gpu_Db.item = new int*[Gpu_Db.sid_len];
    Gpu_Db.iu = new int*[Gpu_Db.sid_len];
    Gpu_Db.ru = new int*[Gpu_Db.sid_len];
    Gpu_Db.tid = new int*[Gpu_Db.sid_len];

    int max_sequence_len=0;

    for(int i=0;i<Gpu_Db.sid_len;i++){
        Gpu_Db.sequence_len[i] = DBdata.sequence_len[i];

        if(Gpu_Db.sequence_len[i]>max_sequence_len){
            max_sequence_len=Gpu_Db.sequence_len[i];
        }

        Gpu_Db.item[i]=DBdata.item[i].data();
        Gpu_Db.iu[i]=DBdata.iu[i].data();
        Gpu_Db.ru[i]=DBdata.ru[i].data();
        Gpu_Db.tid[i]=DBdata.tid[i].data();


//        Gpu_Db.item[i] = new int[Gpu_Db.sequence_len[i]];
//        Gpu_Db.iu[i] = new int[Gpu_Db.sequence_len[i]];
//        Gpu_Db.ru[i] = new int[Gpu_Db.sequence_len[i]];
//        Gpu_Db.tid[i] = new int[Gpu_Db.sequence_len[i]];



        for(int j=0;j<Gpu_Db.sequence_len[i];j++){
            Gpu_Db.item[i][j]=Gpu_Db.DB_item_set_hash[DBdata.item[i][j]];
//            Gpu_Db.iu[i][j]=DBdata.iu[i][j];
//            Gpu_Db.ru[i][j]=DBdata.ru[i][j];
//            Gpu_Db.tid[i][j]=DBdata.tid[i][j];

//            cout<<Gpu_Db.item[i][j]<<" ";
//            cout<<Gpu_Db.iu[i][j]<<" ";
//            cout<<Gpu_Db.ru[i][j]<<" ";
//            cout<<Gpu_Db.tid[i][j]<<"\n";
        }
//        cout<<"\n";

//        cout<<Gpu_Db.sequence_len[i]<<endl;

    }

    Gpu_Db.max_sequence_len=max_sequence_len;

    //Gpu_Db.DB_item_set=DBdata.DB_item_set;


    int item_len=DBdata.single_item_chain.size();

    Gpu_Db.single_item_chain = new int**[item_len];
    Gpu_Db.chain_sid = new int*[item_len]; //每個item出現在哪些sid

    Gpu_Db.c_item_len = item_len;
    Gpu_Db.c_sid_len = new int[item_len]; //每個item的sid長度
    Gpu_Db.c_seq_len = new int*[item_len]; //每個item在不同sid中的seq長度(實例)

    int i_index=0, sid_len, j_index=0,seq_len;
    for(auto i=DBdata.single_item_chain.begin();i!=DBdata.single_item_chain.end();i++){
        sid_len = i->second.size();
        Gpu_Db.single_item_chain[i_index] = new int*[sid_len];
        Gpu_Db.chain_sid[i_index] = new int[sid_len];

        Gpu_Db.c_sid_len[i_index] = sid_len;
        Gpu_Db.c_seq_len[i_index] = new int[sid_len];

        j_index = 0;
        for(auto j = i->second.begin();j!=i->second.end();j++){
            seq_len = j->second.size();
            Gpu_Db.single_item_chain[i_index][j_index] = j->second.data();
            Gpu_Db.chain_sid[i_index][j_index] = j->first;

            Gpu_Db.c_seq_len[i_index][j_index] = seq_len;

            j_index++;
        }
        i_index++;
    }


}


__global__ void PEUcounter(int project_item,
                           int** d_item_project, int** d_iu_project, int** d_ru_project, int** d_tid_project,
                           int* d_sequence_len,int sid_len,
                           int *d_PEU_seq,int *d_Utility_seq) {

    if(threadIdx.x<d_sequence_len[blockIdx.x]){
        if(d_item_project[blockIdx.x][threadIdx.x]==project_item){
            //這裡應該能用redution在加速一點
            atomicMax(&d_PEU_seq[blockIdx.x], d_iu_project[blockIdx.x][threadIdx.x]+d_ru_project[blockIdx.x][threadIdx.x]);
            atomicMax(&d_Utility_seq[blockIdx.x], d_iu_project[blockIdx.x][threadIdx.x]);
        }
    }
}

__global__ void PeuCounter(int project_item,int sid,
                           int** d_item_project, int** d_iu_project, int** d_ru_project, int** d_tid_project,
                           int* d_sequence_len,int sid_len,
                           int *d_PEU_seq,int *d_Utility_seq,int *d_project_point) {
    if(threadIdx.x<d_sequence_len[sid]){
        if(d_item_project[sid][threadIdx.x]==project_item){
            atomicMax(&d_PEU_seq[sid], d_iu_project[sid][threadIdx.x]+d_ru_project[sid][threadIdx.x]);

        }
    }
}

__global__ void Array_add_reduction(int Array_len,int *inputArr,int *outputArr){
    __shared__ int shared_data[1024];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    shared_data[tid] = (idx < Array_len) ? inputArr[idx] : 0;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        outputArr[blockIdx.x] = shared_data[0];
    }
}

__global__ void test(int ***d_single_item_chain,int d_c_item_len,int *d_c_sid_len,int **d_c_seq_len,int **d_chain_sid){
    for(int i=0;i<d_c_item_len;i++){
        printf("item:%d\n",i);
        for(int j=0;j<d_c_sid_len[i];j++){
            printf("sid:%d\n",d_chain_sid[i][j]);
            for(int k=0;k<d_c_seq_len[i][j];k++){
                printf("%d ",d_single_item_chain[i][j][k]);
            }
            printf("\n");
        }
    }
}

void GPUHUSP(const GPU_DB &Gpu_Db){


    size_t freeMem = 0;
    size_t totalMem = 0;

    // 獲取 GPU 的內存信息
    hipError_t status = hipMemGetInfo(&freeMem, &totalMem);

    if (status == hipSuccess) {
        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
    } else {
        cerr << "無法獲取內存信息，錯誤碼: " << hipGetErrorString(status) << endl;
    }

    //project DB初始
    int **d_item_project;
    hipMalloc(&d_item_project, Gpu_Db.sid_len * sizeof(int*));

    int **d_iu_project;
    hipMalloc(&d_iu_project, Gpu_Db.sid_len * sizeof(int*));

    int **d_ru_project;
    hipMalloc(&d_ru_project, Gpu_Db.sid_len * sizeof(int*));

    int **d_tid_project;
    hipMalloc(&d_tid_project, Gpu_Db.sid_len * sizeof(int*));

    // 主機上的指標陣列，用於存放每一行的 d_tmp 指標
    //cudaMemcpy(&d_item_project[i], &d_tmp, sizeof(int*), cudaMemcpyDeviceToDevice);不能這樣,因為d_item_project[i]在host不能讀取
    //要開主機指標的指標（陣列）存Device指標
    int **h_item_project = new int*[Gpu_Db.sid_len];
    int **h_iu_project = new int*[Gpu_Db.sid_len];
    int **h_ru_project = new int*[Gpu_Db.sid_len];
    int **h_tid_project = new int*[Gpu_Db.sid_len];


    for(int i=0;i<Gpu_Db.sid_len;i++){
        int *d_tmp;
        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * sizeof(int));
        hipMemcpy(d_tmp,Gpu_Db.item[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_item_project[i] = d_tmp;

        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * sizeof(int));
        hipMemcpy(d_tmp,Gpu_Db.iu[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_iu_project[i] = d_tmp;

        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * sizeof(int));
        hipMemcpy(d_tmp,Gpu_Db.ru[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_ru_project[i] = d_tmp;

        hipMalloc(&d_tmp, Gpu_Db.sequence_len[i] * sizeof(int));
        hipMemcpy(d_tmp,Gpu_Db.tid[i],Gpu_Db.sequence_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_tid_project[i] = d_tmp;
    }

    hipMemcpy(d_item_project, h_item_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(d_iu_project, h_iu_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(d_ru_project, h_ru_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy(d_tid_project, h_tid_project, Gpu_Db.sid_len * sizeof(int*), hipMemcpyHostToDevice);


    int *d_sequence_len;
    hipMalloc(&d_sequence_len, Gpu_Db.sid_len * sizeof(int));
    hipMemcpy(d_sequence_len,Gpu_Db.sequence_len,Gpu_Db.sid_len * sizeof(int),hipMemcpyHostToDevice);


    //single item chain 初始
    int ***d_single_item_chain;
    hipMalloc((void ***)&d_single_item_chain, Gpu_Db.c_item_len * sizeof(int **));

    int ***h_three_dimension_single_item_chain = new int**[Gpu_Db.c_item_len];//用來存device指標的三維指標
    int **h_two_dimension_single_item_chain;

    int ***h_delete_tmp = new int**[Gpu_Db.c_item_len];//用來cudafree用的

    for(int i=0;i<Gpu_Db.c_item_len;i++){
        h_two_dimension_single_item_chain = new int*[Gpu_Db.c_sid_len[i]];//用來存device指標的二維指標
        for(int j=0;j<Gpu_Db.c_sid_len[i];j++){
            int *d_one_dimension_tmp;
            hipMalloc(&d_one_dimension_tmp,Gpu_Db.c_seq_len[i][j]*sizeof(int));
            hipMemcpy(d_one_dimension_tmp,Gpu_Db.single_item_chain[i][j],Gpu_Db.c_seq_len[i][j]*sizeof(int),hipMemcpyHostToDevice);
            h_two_dimension_single_item_chain[j] = d_one_dimension_tmp;
        }

        int **d_two_dimension_tmp;
        hipMalloc(&d_two_dimension_tmp,Gpu_Db.c_sid_len[i]*sizeof(int*));
        hipMemcpy(d_two_dimension_tmp,h_two_dimension_single_item_chain,Gpu_Db.c_sid_len[i]*sizeof(int*),hipMemcpyHostToDevice);

        h_three_dimension_single_item_chain[i] = d_two_dimension_tmp;
        h_delete_tmp[i]=h_two_dimension_single_item_chain;

        //delete[] h_two_dimension_single_item_chain;
    }
    hipMemcpy(d_single_item_chain,h_three_dimension_single_item_chain,Gpu_Db.c_item_len*sizeof(int**),hipMemcpyHostToDevice);
    //delete[] h_three_dimension_single_item_chain;

    int *d_c_sid_len;
    hipMalloc(&d_c_sid_len,Gpu_Db.c_item_len*sizeof(int));
    hipMemcpy(d_c_sid_len,Gpu_Db.c_sid_len,Gpu_Db.c_item_len*sizeof(int),hipMemcpyHostToDevice);

    int **d_c_seq_len;
    hipMalloc(&d_c_seq_len,Gpu_Db.c_item_len*sizeof(int*));
    int **h_tmp_c_seq_len = new int*[Gpu_Db.c_item_len];

    int **d_chain_sid;
    hipMalloc(&d_chain_sid,Gpu_Db.c_item_len*sizeof(int*));
    int **h_tmp_chain_sid = new int*[Gpu_Db.c_item_len];

    for(int i=0;i<Gpu_Db.c_item_len;i++){
        int *d_tmp;
        hipMalloc(&d_tmp,Gpu_Db.c_sid_len[i]*sizeof(int));
        hipMemcpy(d_tmp,Gpu_Db.c_seq_len[i],Gpu_Db.c_sid_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_tmp_c_seq_len[i] = d_tmp;

        hipMalloc(&d_tmp,Gpu_Db.c_sid_len[i]*sizeof(int));
        hipMemcpy(d_tmp,Gpu_Db.chain_sid[i],Gpu_Db.c_sid_len[i]*sizeof(int),hipMemcpyHostToDevice);
        h_tmp_chain_sid[i] = d_tmp;
    }

    hipMemcpy(d_c_seq_len,h_tmp_c_seq_len,Gpu_Db.c_item_len*sizeof(int*),hipMemcpyHostToDevice);

    hipMemcpy(d_chain_sid,h_tmp_chain_sid,Gpu_Db.c_item_len*sizeof(int*),hipMemcpyHostToDevice);



    int *d_PEU_Arr;//長度是seq數量 用來算peu(可重用)
    hipMalloc(&d_PEU_Arr,Gpu_Db.sid_len*sizeof(int));
    hipMemset(d_PEU_Arr, 0, Gpu_Db.sid_len * sizeof(int));

    int *d_I_list_RSU_Arr,*d_S_list_RSU_Arr;//長度是item數量 用來算rsu(可重用)
    hipMalloc(&d_I_list_RSU_Arr,Gpu_Db.c_item_len*sizeof(int));
    hipMalloc(&d_S_list_RSU_Arr,Gpu_Db.c_item_len*sizeof(int));
    hipMemset(d_I_list_RSU_Arr, 0, Gpu_Db.c_item_len * sizeof(int));
    hipMemset(d_S_list_RSU_Arr, 0, Gpu_Db.c_item_len * sizeof(int));




    test<<<1,1>>>(d_single_item_chain,Gpu_Db.c_item_len,d_c_sid_len,d_c_seq_len,d_chain_sid);

//    for(int i=0;i<Gpu_Db.sid_len;i++){
//        for(int j=0;j<Gpu_Db.sequence_len[i];j++){
//            cout<<Gpu_Db.item[i][j]<<" ";
//        }
//        //cout<<Gpu_Db.sequence_len[i]<<endl;
//        cout<<endl;
//    }

    stack<Tree_node> Main_stack;
    Tree_node top1_Node;

    top1_Node.pattern = "";
    top1_Node.ProjectArr_first_position=new int[Gpu_Db.sid_len];
    top1_Node.ProjectArr_len=new int[Gpu_Db.sid_len];

    //DB clearDBdata;
    //DBdata = clearDBdata;
    //cout<<Gpu_Db.max_sequence_len;










//    for(int i=0;i<Gpu_Db.sid_len;i++){
//        cout<<Gpu_Db.sequence_len[i]<<endl;
//    }
//    cout<<endl;
    //kernelfunction操作

    //如果sequence長度有超過1024需要額外處理
    //int *block_per_sequence = new int[Gpu_Db.sid_len];
//    if(Gpu_Db.max_sequence_len>1024){
//        threadsPerBlock=1024;
//        for(int i=0;i<Gpu_Db.sid_len;i++){
//            blocksPerGrid+= (Gpu_Db.sequence_len[i]+threadsPerBlock-1)/threadsPerBlock;
//            block_per_sequence[i] = (Gpu_Db.sequence_len[i]+threadsPerBlock-1)/threadsPerBlock;
//        }
//    }else{
//        threadsPerBlock=Gpu_Db.max_sequence_len;
//        blocksPerGrid=Gpu_Db.sid_len;
//    }

    int *d_PEU_seq;
    hipMalloc(&d_PEU_seq, Gpu_Db.sid_len * sizeof(int));
    hipMemset(d_PEU_seq, 0, Gpu_Db.sid_len * sizeof(int));

    int d_PEU_add_len = (Gpu_Db.sid_len + 1024 - 1) / 1024;//看有多少seq在用1024切
    int *d_PEU_add_result;
    hipMalloc(&d_PEU_add_result, d_PEU_add_len * sizeof(int));

    int *d_Utility_seq;
    hipMalloc(&d_Utility_seq, Gpu_Db.sid_len * sizeof(int));
    hipMemset(d_Utility_seq, 0, Gpu_Db.sid_len * sizeof(int));



    int threadsPerBlock;
    int blocksPerGrid;



    for(int i:Gpu_Db.DB_item_set){
        threadsPerBlock=Gpu_Db.max_sequence_len;
        blocksPerGrid=Gpu_Db.sid_len;
//        for(int j=0;j<Gpu_Db.sid_len;j++){
//            PeuCounter<<<1,Gpu_Db.sequence_len[j]>>>(i,j,d_item_project,d_iu_project,d_ru_project,d_tid_project
//                    ,d_sequence_len,Gpu_Db.sid_len,d_PEU_seq,d_Utility_seq,d_project_point);
//        }

        //一次開好比較快
        PEUcounter<<<blocksPerGrid,threadsPerBlock>>>(i,d_item_project,d_iu_project,d_ru_project,d_tid_project
                ,d_sequence_len,Gpu_Db.sid_len,d_PEU_seq,d_Utility_seq);

//        int *h_PEU_seq=new int[Gpu_Db.sid_len];
//        cudaMemcpy(h_PEU_seq, d_PEU_seq, Gpu_Db.sid_len*sizeof(int), cudaMemcpyDeviceToHost);
//
//        //這裡可以用加總
//        int PEU_count=0;
//        for(int j=0;j<Gpu_Db.sid_len;j++){
//            PEU_count+=h_PEU_seq[j];
//        }
//        cout<<"item:"<<i<<endl;
//        cout<<"PEU:"<<PEU_count<<endl<<endl;
        //
//        int *h_Utility_seq=new int[Gpu_Db.sid_len];
//        cudaMemcpy(h_Utility_seq, d_Utility_seq, Gpu_Db.sid_len*sizeof(int), cudaMemcpyDeviceToHost);


        Array_add_reduction<<<d_PEU_add_len,1024>>>(Gpu_Db.sid_len,d_PEU_seq,d_PEU_add_result);
        int *h_PEU_add_result=new int[d_PEU_add_len];
        hipMemcpy(h_PEU_add_result, d_PEU_add_result, d_PEU_add_len*sizeof(int), hipMemcpyDeviceToHost);

        int PEU_count = 0;
        for (int j = 0; j < d_PEU_add_len; j++) {
            PEU_count += h_PEU_add_result[j];
        }
//        cout<<"item:"<<i<<endl;
//        cout<<"PEU:"<<PEU_count<<endl;





//        if(PEU_count<threshold){
//            continue;
//        }

        hipMemset(d_PEU_seq, 0, Gpu_Db.sid_len * sizeof(int));
        //cout<<"*/*****\n";
    }



//    PEUcounter<<<blocksPerGrid,threadsPerBlock>>>(821024,d_item_project,d_iu_project,d_ru_project,d_tid_project,d_sequence_len,Gpu_Db.sid_len,d_PEU_seq);
//
//    int *h_Result=new int[Gpu_Db.sid_len];
//    cudaMemcpy(h_Result, d_PEU_seq, Gpu_Db.sid_len*sizeof(int), cudaMemcpyDeviceToHost);
//
//    for(int j=0;j<Gpu_Db.sid_len;j++){
//        cout<<h_Result[j]<<endl;
//    }

    //project free
    hipFree(d_item_project);
    hipFree(d_iu_project);
    hipFree(d_ru_project);
    hipFree(d_tid_project);

    for(int i=0;i<Gpu_Db.sid_len;i++) {
        hipFree(h_item_project[i]);
        hipFree(h_iu_project[i]);
        hipFree(h_ru_project[i]);
        hipFree(h_tid_project[i]);
    }

    delete[] h_item_project;
    delete[] h_iu_project;
    delete[] h_ru_project;
    delete[] h_tid_project;
    //delete[] h_Result;

    //chain free
    hipFree(d_single_item_chain);

    for(int i=0;i<Gpu_Db.c_item_len;i++){
        hipFree(h_three_dimension_single_item_chain[i]);
        for(int j=0;j<Gpu_Db.c_sid_len[i];j++){
            hipFree(h_delete_tmp[i][j]);
        }
    }

    delete[] h_three_dimension_single_item_chain;
    delete[] h_delete_tmp;

    hipFree(d_c_seq_len);
    hipFree(d_chain_sid);

    for(int i=0;i<Gpu_Db.c_item_len;i++){
        hipFree(h_tmp_c_seq_len[i]);
        hipFree(h_tmp_chain_sid[i]);
    }

    delete [] h_tmp_c_seq_len;
    delete [] h_tmp_chain_sid;


    hipFree(d_sequence_len);
    hipFree(d_PEU_seq);

    // 獲取 GPU 的內存信息
    status = hipMemGetInfo(&freeMem, &totalMem);

    if (status == hipSuccess) {
        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
    } else {
        cerr << "無法獲取內存信息，錯誤碼: " << hipGetErrorString(status) << endl;
    }
//    // 獲取 GPU 的內存信息
//    status = cudaMemGetInfo(&freeMem, &totalMem);
//
//    if (status == cudaSuccess) {
//        cout << "GPU 總內存: " << totalMem / (1024 * 1024) << " MB" << endl;
//        cout << "GPU 可用內存: " << freeMem / (1024 * 1024) << " MB" << endl;
//    } else {
//        cerr << "無法獲取內存信息，錯誤碼: " << cudaGetErrorString(status) << endl;
//    }


//    for(int i=0;i<Gpu_Db.sid_len;i++) {
//        for (int j = 0; j < Gpu_Db.sequence_len[i]; j++) {
//            cout << Gpu_Db.item[i][j] << " ";
//            cout << Gpu_Db.iu[i][j] << " ";
//            cout << Gpu_Db.ru[i][j] << " ";
//            cout << Gpu_Db.tid[i][j] << "\n";
//        }
//        cout << Gpu_Db.sequence_len[i] << endl;
//    }

}


int main() {
    // 指定要讀取的檔案名稱
    string filename = "YoochooseSamller.txt";
    ifstream file(filename);
    vector<string> lines;

    // 檢查檔案是否成功開啟
    if (!file.is_open()) {
        cerr << "無法開啟檔案: " << filename << endl;
        return 1; // 返回錯誤代碼
    }


    DB DBdata;
    parseData(file,DBdata);

//    cout<<test_max_seq;
    file.close(); // 關閉檔案

    double threshold = 0.01 * DBdata.DButility;

    SWUpruning(threshold,DBdata);

    GPU_DB Gpu_Db;

    auto start = std::chrono::high_resolution_clock::now();
    Bulid_GPU_DB(DBdata,Gpu_Db);
    auto end = std::chrono::high_resolution_clock::now();
    // 計算持續時間，並轉換為毫秒
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds" << std::endl;

//    for(int i=0;i<Gpu_Db.c_item_len;i++){
//        for(int j=0;j<Gpu_Db.c_sid_len[i];j++){
//            cout<<Gpu_Db.chain_sid[i][j]<<"\n";
//            for(int k=0;k<Gpu_Db.c_seq_len[i][j];k++){
//                cout<<Gpu_Db.single_item_chain[i][j][k]<<" "<<endl;
//            }
//        }
//    }



    GPUHUSP(Gpu_Db);

    return 0;
}
